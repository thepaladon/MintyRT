#include "hip/hip_runtime.h"
﻿// GLM Defines
#define CUDA_VERSION 12020
#define GLM_FORCE_CUDA

#include <chrono>
#include <cstdlib>

#include "hip/hip_runtime.h"
#include ""
#include "Ray.cuh"

#include "Window.h"
#include "ModelLoading/Buffer.h"
#include "ModelLoading/Model.h"

#include "Camera.cuh"
#include "CudaUtils.cuh"

constexpr int FB_INIT_WIDTH = 1200; 
constexpr int FB_INIT_HEIGHT= 800;

#define MODEL_FP(model) (std::string("Resources/Models/") + model + "/" + model + ".gltf")

__device__ struct Triangle
{
public:
    glm::vec3 vertex0;
    glm::vec3 vertex1;
    glm::vec3 vertex2;
};

__device__ bool intersect_tri(Ray& ray, const Triangle& tris)
{
    const glm::vec3 edge1 = tris.vertex1 - tris.vertex0;
    const glm::vec3 edge2 = tris.vertex2 - tris.vertex0;
    const glm::vec3 h = cross(ray.d, edge2);
    const float a = dot(edge1, h);
    if (fabs(a) < 0.0001) return false; // ray parallel to triangle
    const float f = 1 / a;
    const glm::vec3 s = ray.o - tris.vertex0;
    const float u = f * dot(s, h);
    if (u < 0 || u > 1) return false;
    const glm::vec3 q = cross(s, edge1);
    const float v = f * dot(ray.d, q);
    if (v < 0 || u + v > 1) return false;
    const float t = f * dot(edge2, q);
    if (t > 0.0001f) {
        if (ray.t > t)
        {
            ray.t = t;
            //ray->intersection.tri_hit = triIdx;
            //ray->intersection.u = u;
            //ray->intersection.v = v;
            //ray->intersection.header_tri_count = header[0].tris_count;
            //ray->intersection.geo_normal = cross(edge1, edge2);
        }
        return true;
    }
    return false;
}

__device__ glm::vec3 color(Ray& r, const float* vertex, const unsigned* idx, unsigned long long  num_tris) {

    for (int i = 0; i < num_tris; i++)
    {
        const auto& i0 = idx[i * 3 + 0];
        const auto& i1 = idx[i * 3 + 1];
        const auto& i2 = idx[i * 3 + 2];
        
        const auto& v0x = vertex[i0 * 3 + 0];
        const auto& v0y = vertex[i0 * 3 + 1];
        const auto& v0z = vertex[i0 * 3 + 2];

    	const auto& v1x = vertex[i1 * 3 + 0];
        const auto& v1y = vertex[i1 * 3 + 1];
        const auto& v1z = vertex[i1 * 3 + 2];

        const auto& v2x = vertex[i2 * 3 + 0];
        const auto& v2y = vertex[i2 * 3 + 1];
        const auto& v2z = vertex[i2 * 3 + 2];

        const glm::vec3& v0 = glm::vec3(v0x, v0y, v0z);
        const glm::vec3& v1 = glm::vec3(v1x, v1y, v1z);
        const glm::vec3& v2 = glm::vec3(v2x, v2y, v2z);
        
        Triangle tri{ v0, v1, v2 };

        if (intersect_tri(r, tri))
        {
            return { 1.0f, 0.0f, 0.0f };
        }
    }

    glm::vec3 unit_direction = normalize(r.direction());
    float t = 0.5f * (unit_direction.y + 1.0f);
    return (1.0f - t) * glm::vec3(1.0f, 1.0f, 1.0f) + t * glm::vec3(0.5f, 0.7f, 1.0f);
}

__global__ void render(uchar3* fb, int max_x, int max_y, Camera cam, const float* vertex, const unsigned* idx, unsigned long long num_tris) {

    const int i = threadIdx.x + blockIdx.x * blockDim.x;
    const int j = threadIdx.y + blockIdx.y * blockDim.y;
    if ((i >= max_x) || (j >= max_y)) return;

    int pixel_index = j * max_x + i;
 	Ray r = cam.generate((float)max_x, (float)max_y, (float)i, (float)j);
    fb[pixel_index] = to_uchar3(color(r, vertex, idx, num_tris));
}


int main()
{
	auto* m_Window = new Window(FB_INIT_WIDTH, FB_INIT_HEIGHT, "Minty Cuda RT");

    uchar3* gpu_fb;
    uchar3* cpu_fb = nullptr;

    uint32_t alignedX = m_Window->GetAlignedWidth();
    uint32_t alignedY = m_Window->GetAlignedHeight();

	// Initial Allocate Frame Buffer
	{
        const int num_pixels = alignedX * alignedY;
        const size_t fb_size = num_pixels * sizeof(uchar3);
        checkCudaErrors(hipMallocManaged((void**)&gpu_fb, fb_size));
        cpu_fb = new uchar3[num_pixels];
    }

	// const auto truck = new bml::Model(MODEL_FP("CesiumMilkTruck"));
	// const auto dmged_helm = new bml::Model(MODEL_FP("DamagedHelmet"));
	const auto sahhhduh = new bml::Model(MODEL_FP("sah_test"));
	// const auto scifi_helm = new bml::Model(MODEL_FP("SciFiHelmet"));

    // Start the timer
    auto start_time = std::chrono::high_resolution_clock::now();
	auto end_time = std::chrono::high_resolution_clock::now();
    float run_timer_s = 0.0f;

    // Output FB
    bool running = true;

	Camera cam(glm::vec3(0.0f, 0.0f, 0.0f), glm::vec3(0.0f, glm::radians(-224.f), 0.0f), 75.f, float(alignedX) / float(alignedY));

    glm::vec3 v0 = glm::vec3(1.0f, 1.0f, 1.0f);
    glm::vec3 v1 = glm::vec3(1.0f, 0.0f, 1.0f);
    glm::vec3 v2 = glm::vec3(0.0f, 1.0f, 1.0f);
    glm::vec3 v3 = glm::vec3(0.0f, 0.0f, 1.0f);
    
    bml::Buffer* vert_buff = nullptr;
    bml::Buffer* idx_buff = nullptr;
    {

        glm::vec3 vert[4] = 
            { v0, v1, v2, v3 };

        /*
        glm::vec3 quad[4] = {
            { v3, v1, v2 },
            { v0, v1, v2 } };
        */

        int idx[6] = { 3, 1, 2, 2, 1, 0};

        vert_buff = new bml::Buffer(vert, sizeof(glm::vec3), 4, "Vertex Buffer");
        idx_buff = new bml::Buffer(idx, sizeof(int), 6, "Idx Buffer");
    }

    //Make sure everything is available before start of Render
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());

    // const void* vrtx_buffer = vert_buff->GetBufferDataPtr();
    // const void* idx_buffer = idx_buff->GetBufferDataPtr();
    // const unsigned long long num_tris = 2;

 	const void* vrtx_buffer = sahhhduh->GetBuffers()[0]->GetBufferDataPtr();
 	const void* idx_buffer = sahhhduh->GetBuffers()[3]->GetBufferDataPtr();
    const unsigned long long num_tris = sahhhduh->GetBuffers()[0]->GetNumElements() / 3;

    while (running)
    {
        // Note, resizing and moving the window won't be caught in DT because it happens in m_Window->Update()
        // This is desired behavior because nobody likes things to jump around 
        end_time = std::chrono::high_resolution_clock::now();
        std::chrono::duration<float> delta_time_s = end_time - start_time; // in seconds
        run_timer_s += delta_time_s.count();
        float delta_time_ms = delta_time_s.count() * 1000;
    	start_time = std::chrono::high_resolution_clock::now();

        // Replace with mouse controls once that is implemented in a good way.
        float hor_inp = 0;
        float ver_inp = 0;
        if (m_Window->GetKey(VK_LEFT)) { hor_inp = -1.0; }
        if (m_Window->GetKey(VK_RIGHT)) { hor_inp = 1.0; }
        if (m_Window->GetKey(VK_UP)) { ver_inp = -1.0;   }
        if (m_Window->GetKey(VK_DOWN)) { ver_inp = 1.0;  }

        const float m_dtx = hor_inp ; // m_Window->GetMouseDeltaX();
        const float m_dty = ver_inp ; // m_Window->GetMouseDeltaY();

        cam.dt = delta_time_ms;
        if (m_Window->GetKey('W'))
        {
            cam.MoveFwd(1.0f);
        }

        if (m_Window->GetKey('S'))
        {
            cam.MoveFwd(-1.0f);
        }

        if (m_Window->GetKey('D'))
        {
            cam.MoveRight(1.0f);
        }

        if (m_Window->GetKey('A'))
        {
            cam.MoveRight(-1.0f);
        }

        if (m_Window->GetKey('R'))
        {
            cam.MoveUp(1.0f);
        }

    	if (m_Window->GetKey('F'))
        {
            cam.MoveUp(-1.0f);
        }

        cam.SetPitch(m_dty);
        cam.SetYaw(m_dtx);
    	cam.UpdateCamera();

        //printf(" %f          %f \n", m_dtx, m_dty );
        //printf("Pos - X: %f, Y: %f, Z : %f \n", cam.m_Pos.x, cam.m_Pos.y, cam.m_Pos.z );
        //auto rad = glm::degrees(cam.m_PitchYawRoll);
    	//printf("Pitch: %f, Yaw: %f, Roll: %f \n \n", rad.x, rad.y, rad.z );

        running = m_Window->OnUpdate(delta_time_ms);

        if (m_Window->GetIsResized()) {
            m_Window->CreateSampleDIB();

            alignedX = m_Window->GetAlignedWidth();
            alignedY = m_Window->GetAlignedHeight();

        	checkCudaErrors(hipFree(gpu_fb));
            delete cpu_fb;

            const int num_pixels = alignedX * alignedY;
            const size_t fb_size = num_pixels * sizeof(uchar3);
            checkCudaErrors(hipMallocManaged((void**)&gpu_fb, fb_size));
            cpu_fb = new uchar3[alignedX * alignedY];

            printf("Resized : %i : %i \n", alignedX, alignedY);
        }

        // Thread Groups
        constexpr int tx = 8;
        constexpr int ty = 8;


        // Render our buffer
        const dim3 blocks(alignedX / tx + 1, alignedY / ty + 1);
        const dim3 threads(tx, ty);
        render <<< blocks, threads >>> (
            gpu_fb, 
            alignedX, 
            alignedY, 
            cam,
            (const float*)(vrtx_buffer),
            (const unsigned*)(idx_buffer),
            num_tris
            );



        checkCudaErrors(hipGetLastError());
        checkCudaErrors(hipDeviceSynchronize());

        hipMemcpy(cpu_fb, gpu_fb, alignedX * alignedY * sizeof(uchar3), hipMemcpyDeviceToHost);

        m_Window->RenderFb(cpu_fb);
    }

    m_Window->Shutdown();
    delete m_Window;
    delete cpu_fb;
    checkCudaErrors(hipFree(gpu_fb));

    return 0;
}


