#include "hip/hip_runtime.h"
﻿
#include <chrono>
#include <cstdlib>
#include <iostream>

#include "Camera.cuh"
#include "hip/hip_runtime.h"
#include ""
#include "Ray.cuh"
#include "Vec3.cuh"

#include "Window.h"

constexpr int FB_WIDTH = 943; 
constexpr int FB_HEIGHT= 540;

#define checkCudaErrors(val) check_cuda( (val), #val, __FILE__, __LINE__ )
void check_cuda(hipError_t result, char const* const func, const char* const file, int const line) {
    if (result) {
        std::cerr << "CUDA error = " << static_cast<unsigned int>(result) << " at " <<
            file << ":" << line << " '" << func << "' \n";
        // Make sure we call CUDA Device Reset before exiting
        hipDeviceReset();
        exit(99);
    }
}

__device__ Vec3 color(const Ray& r) {
   Vec3 unit_direction = r.direction().normalize();
   float t = 0.5f * (unit_direction.y() + 1.0f);
   return (1.0f - t) * Vec3(1.0, 1.0, 1.0) + t * Vec3(0.5, 0.7, 1.0);
}


__global__ void render(uchar3* fb, int max_x, int max_y, Camera cam) {
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    int j = threadIdx.y + blockIdx.y * blockDim.y;
    if ((i >= max_x) || (j >= max_y)) return;
    int pixel_index = j * max_x + i;
    float u = float(i) / float(max_x);
    float v = float(j) / float(max_y);
    Ray r = cam.generate((float)max_x, (float)max_y, u, v);
    fb[pixel_index] = color(r).to_uchar3();
}



int main()
{
	auto* m_Window = new Window(FB_WIDTH, FB_HEIGHT, "Minty Cuda RT");

    uchar3* gpu_fb;
    uchar3* cpu_fb = nullptr;

    uint32_t alignedX = m_Window->GetAlignedWidth();
    uint32_t alignedY = m_Window->GetAlignedHeight();

	// Initial Allocate Frame Buffer
	{
        const int num_pixels = alignedX * alignedY;
        const size_t fb_size = num_pixels * sizeof(uchar3);
        checkCudaErrors(hipMallocManaged((void**)&gpu_fb, fb_size));
        cpu_fb = new uchar3[num_pixels];
    }

    
    // Start the timer
    auto start_time = std::chrono::high_resolution_clock::now();
	auto end_time = std::chrono::high_resolution_clock::now();
    float run_timer_s = 0.0f;

    // Output FB
    bool running = true;
    while (running)
    {
        // Note, resizing and moving the window won't be caught in DT because it happens in m_Window->Update()
        // This is desired behavior because nobody likes things to jump around 
        end_time = std::chrono::high_resolution_clock::now();
        std::chrono::duration<float> delta_time_s = end_time - start_time; // in seconds
        run_timer_s += delta_time_s.count();
    	start_time = std::chrono::high_resolution_clock::now();


        running = m_Window->OnUpdate();

        if (m_Window->GetIsResized()) {
            m_Window->CreateSampleDIB();

            alignedX = m_Window->GetAlignedWidth();
            alignedY = m_Window->GetAlignedHeight();

        	checkCudaErrors(hipFree(gpu_fb));
            delete cpu_fb;

            const int num_pixels = alignedX * alignedY;
            const size_t fb_size = num_pixels * sizeof(uchar3);
            checkCudaErrors(hipMallocManaged((void**)&gpu_fb, fb_size));
            cpu_fb = new uchar3[alignedX * alignedY];

            printf("Resized : %i : %i \n", alignedX, alignedY);
        }

        // Thread Groups
        int tx = 8;
        int ty = 8;

        const Camera cam(Vec3(0.0f), Vec3(0.0f, 1.0f, 0.0f), Vec3(0.f, -0.6f, -1.f), 90.0f, float(alignedX) / float(alignedY));

        // Render our buffer
        dim3 blocks(alignedX / tx + 1, alignedY / ty + 1);
        dim3 threads(tx, ty);
        render << <blocks, threads >> > (
            gpu_fb, 
            alignedX, 
            alignedY, 
            cam
            );
        checkCudaErrors(hipGetLastError());
        checkCudaErrors(hipDeviceSynchronize());

        hipMemcpy(cpu_fb, gpu_fb, alignedX * alignedY * sizeof(uchar3), hipMemcpyDeviceToHost);

        m_Window->RenderFb(cpu_fb);
    }

    m_Window->Shutdown();
    delete m_Window;
    delete cpu_fb;
    checkCudaErrors(hipFree(gpu_fb));

    return 0;
}


