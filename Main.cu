#include "hip/hip_runtime.h"
﻿// GLM Defines
#define CUDA_VERSION 12020
#define GLM_FORCE_CUDA

#include <chrono>
#include <cstdlib>
#include <iostream>

#include "Camera.cuh"
#include "hip/hip_runtime.h"
#include ""
#include "Ray.cuh"
#include "Utils.h"

#include "Window.h"


constexpr int FB_WIDTH = 1200; 
constexpr int FB_HEIGHT= 800;

#define checkCudaErrors(val) check_cuda( (val), #val, __FILE__, __LINE__ )
void check_cuda(hipError_t result, char const* const func, const char* const file, int const line) {
    if (result) {
        std::cerr << "CUDA error = " << static_cast<unsigned int>(result) << " at " <<
            file << ":" << line << " '" << func << "' \n";
        // Make sure we call CUDA Device Reset before exiting
        hipDeviceReset();
        exit(99);
    }
}

struct Triangle
{
public:
    glm::vec3 vertex0;
    glm::vec3 vertex1;
    glm::vec3 vertex2;
};

 __device__ bool intersect_tri(Ray& ray, const Triangle* tris, glm::uint triIdx)
{
    const glm::vec3 edge1 = tris[triIdx].vertex1 - tris[triIdx].vertex0;
    const glm::vec3 edge2 = tris[triIdx].vertex2 - tris[triIdx].vertex0;
    const glm::vec3 h = cross(ray.d, edge2);
    const float a = dot(edge1, h);
    if (fabs(a) < 0.0001) return false; // ray parallel to triangle
    const float f = 1 / a;
    const glm::vec3 s = ray.o - tris[triIdx].vertex0;
    const float u = f * dot(s, h);
    if (u < 0 || u > 1) return false;
    const glm::vec3 q = cross(s, edge1);
    const float v = f * dot(ray.d, q);
    if (v < 0 || u + v > 1) return false;
    const float t = f * dot(edge2, q);
    if (t > 0.0001f)
    if (ray.t > t)
    {
        ray.t = t;
        //ray->intersection.tri_hit = triIdx;
        //ray->intersection.u = u;
        //ray->intersection.v = v;
        //ray->intersection.header_tri_count = header[0].tris_count;
        //ray->intersection.geo_normal = cross(edge1, edge2);
    }
    return true;
}






__device__ glm::vec3 color(Ray& r) {

    glm::vec3 v0 = glm::vec3(0.0f, 1.0f, 1.0f);
    glm::vec3 v1 = glm::vec3(.5f, 0.0f, 1.0f);
    glm::vec3 v2 = glm::vec3(-.5f, 0.0f, 1.0f);

    Triangle tri{ v0, v1, v2 } ;

    glm::vec3 point(99000.0f);
    glm::vec3 normal(420.420f);
    float t;


    if (intersect_tri(r, &tri, 0))
    {
    	return { 1.0f, 0.0f, 0.0f };
    }
    else {
        glm::vec3 unit_direction = normalize(r.direction());
        float t = 0.5f * (unit_direction.y + 1.0f);
        return (1.0f - t) * glm::vec3(1.0f, 1.0f, 1.0f) + t * glm::vec3(0.5f, 0.7f, 1.0f);

    }
}

__global__ void render(uchar3* fb, int max_x, int max_y, Camera cam) {

    const int i = threadIdx.x + blockIdx.x * blockDim.x;
    const int j = threadIdx.y + blockIdx.y * blockDim.y;
    if ((i >= max_x) || (j >= max_y)) return;
    int pixel_index = j * max_x + i;
    Ray r = cam.generate((float)max_x, (float)max_y, (float)i, (float)j);
    fb[pixel_index] = to_uchar3(color(r));
}



int main()
{
	auto* m_Window = new Window(FB_WIDTH, FB_HEIGHT, "Minty Cuda RT");

    uchar3* gpu_fb;
    uchar3* cpu_fb = nullptr;

    uint32_t alignedX = m_Window->GetAlignedWidth();
    uint32_t alignedY = m_Window->GetAlignedHeight();

	// Initial Allocate Frame Buffer
	{
        const int num_pixels = alignedX * alignedY;
        const size_t fb_size = num_pixels * sizeof(uchar3);
        checkCudaErrors(hipMallocManaged((void**)&gpu_fb, fb_size));
        cpu_fb = new uchar3[num_pixels];
    }

    // Start the timer
    auto start_time = std::chrono::high_resolution_clock::now();
	auto end_time = std::chrono::high_resolution_clock::now();
    float run_timer_s = 0.0f;

    // Output FB
    bool running = true;

	Camera cam(glm::vec3(0.0f, 0.0f, 0.0f), glm::vec3(0.0f, glm::radians(-224.f), 0.0f), 75.f, float(alignedX) / float(alignedY));

    while (running)
    {
        // Note, resizing and moving the window won't be caught in DT because it happens in m_Window->Update()
        // This is desired behavior because nobody likes things to jump around 
        end_time = std::chrono::high_resolution_clock::now();
        std::chrono::duration<float> delta_time_s = end_time - start_time; // in seconds
        run_timer_s += delta_time_s.count();
    	start_time = std::chrono::high_resolution_clock::now();


        const float sensitivity = .5f;

        float hor_inp = 0;
        float ver_inp = 0;
        if (m_Window->GetKey(VK_LEFT)) { hor_inp = 1.0; }
        if (m_Window->GetKey(VK_RIGHT)) { hor_inp = -1.0; }
        if (m_Window->GetKey(VK_UP)) { ver_inp = 1.0; }
        if (m_Window->GetKey(VK_DOWN)) { ver_inp = -1.0; }


        float m_dtx = hor_inp ;// m_Window->GetMouseDeltaX();
        float m_dty = ver_inp ;// m_Window->GetMouseDeltaY();


        if (m_Window->GetKey('W'))
        {
            cam.MoveFwd(1.0f);
        }

        if (m_Window->GetKey('S'))
        {
            cam.MoveFwd(-1.0f);
        }

        if (m_Window->GetKey('D'))
        {
            cam.MoveRight(1.0f);
        }

        if (m_Window->GetKey('A'))
        {
            cam.MoveRight(-1.0f);
        }

        if (m_Window->GetKey('R'))
        {
            cam.MoveUp(1.0f);
        }

    	if (m_Window->GetKey('F'))
        {
            cam.MoveUp(-1.0f);
        }

        cam.SetPitch(m_dty);
        cam.SetYaw(m_dtx);
    	cam.UpdateCamera();

        //printf(" %f          %f \n", m_dtx, m_dty );

        
        printf("Pos - X: %f, Y: %f, Z : %f \n", cam.m_Pos.x, cam.m_Pos.y, cam.m_Pos.z );

        auto rad = glm::degrees(cam.m_PitchYawRoll);
    	printf("Pitch: %f, Yaw: %f, Roll: %f \n \n", rad.x, rad.y, rad.z );

        running = m_Window->OnUpdate();

        if (m_Window->GetIsResized()) {
            m_Window->CreateSampleDIB();

            alignedX = m_Window->GetAlignedWidth();
            alignedY = m_Window->GetAlignedHeight();

        	checkCudaErrors(hipFree(gpu_fb));
            delete cpu_fb;

            const int num_pixels = alignedX * alignedY;
            const size_t fb_size = num_pixels * sizeof(uchar3);
            checkCudaErrors(hipMallocManaged((void**)&gpu_fb, fb_size));
            cpu_fb = new uchar3[alignedX * alignedY];

            printf("Resized : %i : %i \n", alignedX, alignedY);
        }

        // Thread Groups
        int tx = 8;
        int ty = 8;

        
        // Render our buffer
        dim3 blocks(alignedX / tx + 1, alignedY / ty + 1);
        dim3 threads(tx, ty);
        render << <blocks, threads >> > (
            gpu_fb, 
            alignedX, 
            alignedY, 
            cam
            );
        checkCudaErrors(hipGetLastError());
        checkCudaErrors(hipDeviceSynchronize());

        hipMemcpy(cpu_fb, gpu_fb, alignedX * alignedY * sizeof(uchar3), hipMemcpyDeviceToHost);

        m_Window->RenderFb(cpu_fb);
    }

    m_Window->Shutdown();
    delete m_Window;
    delete cpu_fb;
    checkCudaErrors(hipFree(gpu_fb));

    return 0;
}


