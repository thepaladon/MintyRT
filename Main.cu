#include "hip/hip_runtime.h"
﻿
#include <cstdlib>
#include <iostream>

#include "hip/hip_runtime.h"
#include ""

#include "Window.h"



constexpr int FB_WIDTH = 943; 
constexpr int FB_HEIGHT= 540;

#define checkCudaErrors(val) check_cuda( (val), #val, __FILE__, __LINE__ )
void check_cuda(hipError_t result, char const* const func, const char* const file, int const line) {
    if (result) {
        std::cerr << "CUDA error = " << static_cast<unsigned int>(result) << " at " <<
            file << ":" << line << " '" << func << "' \n";
        // Make sure we call CUDA Device Reset before exiting
        hipDeviceReset();
        exit(99);
    }
}

__global__ void render(uchar3* fb, int max_x, int max_y) {
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    int j = threadIdx.y + blockIdx.y * blockDim.y;
    if ((i >= max_x) || (j >= max_y)) return;
    int pixel_index = j * max_x + i ;
    fb[pixel_index].z = i * 255 / max_x;
    fb[pixel_index].y = j * 255 / max_y;
    fb[pixel_index].x = 10;
}


int main()
{
	auto* m_Window = new Window(FB_WIDTH, FB_HEIGHT, "Minty Cuda RT");

    uchar3* gpu_fb;
    uchar3* cpu_fb = nullptr;

    uint32_t alignedX = m_Window->GetAlignedWidth();
    uint32_t alignedY = m_Window->GetAlignedHeight();

	// Initial Allocate Frame Buffer
	{
        const int num_pixels = alignedX * alignedY;
        const size_t fb_size = num_pixels * sizeof(uchar3);
        checkCudaErrors(hipMallocManaged((void**)&gpu_fb, fb_size));
        cpu_fb = new uchar3[num_pixels];
    }


    // Output FB
    bool running = true;
    while (running)
    {
        running = m_Window->OnUpdate();

        if (m_Window->GetIsResized()) {
            m_Window->CreateSampleDIB();

            alignedX = m_Window->GetAlignedWidth();
            alignedY = m_Window->GetAlignedHeight();

        	checkCudaErrors(hipFree(gpu_fb));
            delete cpu_fb;

            const int num_pixels = alignedX * alignedY;
            const size_t fb_size = num_pixels * sizeof(uchar3);
            checkCudaErrors(hipMallocManaged((void**)&gpu_fb, fb_size));
            cpu_fb = new uchar3[alignedX * alignedY];

            printf("Resized : %i : %i \n", alignedX, alignedY);
        }

        // Thread Groups
        int tx = 8;
        int ty = 8;

        // Render our buffer
        dim3 blocks(alignedX / tx + 1, alignedY / ty + 1);
        dim3 threads(tx, ty);
        render << <blocks, threads >> > (gpu_fb, alignedX, alignedY);
        checkCudaErrors(hipGetLastError());
        checkCudaErrors(hipDeviceSynchronize());

        hipMemcpy(cpu_fb, gpu_fb, alignedX * alignedY * sizeof(uchar3), hipMemcpyDeviceToHost);

        m_Window->RenderFb(cpu_fb);
    }


    m_Window->Shutdown();
    delete m_Window;
    delete cpu_fb;
    checkCudaErrors(hipFree(gpu_fb));

    return 0;
}


